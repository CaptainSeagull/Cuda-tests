// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b) {
    a[threadIdx.x] += b[threadIdx.x];
}

void *my_malloc(size_t size) {
    void *p;
    hipMalloc(&p, size);

    // TODO(Jonny): Check error here! If there is an error, free the memory and return null.

    return(p);
}

void my_free(void *p) {
    if(!p) { assert(0); }
    else {
        hipFree(p);
        // TODO(Jonny): Check error here!
    }
}

// TODO(Jonny): Do cudaMemcpy!

int main() {
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    ad = (char *)my_malloc(csize);
    bd = (int *)my_malloc(isize);
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello <<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
    my_free(ad);
    my_free(bd);


    printf("%s\n", a);
    return EXIT_SUCCESS;
}